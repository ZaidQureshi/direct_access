#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdint>
#include <iostream>
#include <chrono>
#include "kernels.hu"

#define BLK_SIZE (128)
#define GRID_SIZE (1024ULL)
#define N_BLKS  (GRID_SIZE/BLK_SIZE)

typedef ulong4 d_t;

enum bench_type { READ = 0, WRITE = 1, MIXED = 2};

int main(int argc, char *argv[]) {
    if (argc < 4) {
        std::cerr << "Please specify GPU id, number of elems, and bench type\n";
        exit(1);

    }


    const unsigned int gpu_id = std::stoul(std::string(argv[1]));
    const unsigned int num_elems = std::stoul(std::string(argv[2]));
    const unsigned int type = std::stoul(std::string(argv[3]));

    cuda_err_chk(hipSetDevice(gpu_id));

    hipSetDeviceFlags(hipDeviceMapHost);

    d_t * h_arr = nullptr;
    d_t * d_flag = nullptr;


    cuda_err_chk(hipHostAlloc((void **)&h_arr,  num_elems * sizeof(d_t),  hipHostMallocMapped));
    cuda_err_chk(hipMalloc((void **) &d_flag, sizeof(d_t)));

    d_t * d_arr = nullptr;

    cuda_err_chk(hipHostGetDevicePointer((void **)&d_arr,  (void *) h_arr , 0));

    auto start = std::chrono::high_resolution_clock::now();
    switch (type) {
    case READ:
        gpu_read<BLK_SIZE, d_t><<<GRID_SIZE,BLK_SIZE>>>(d_arr, d_flag, num_elems * sizeof(d_t));
        break;
    case WRITE:
        gpu_write<BLK_SIZE, d_t><<<GRID_SIZE,BLK_SIZE>>>(d_arr, num_elems * sizeof(d_t));
        break;
    case MIXED:
        gpu_mix<BLK_SIZE, d_t><<<GRID_SIZE,BLK_SIZE>>>(d_arr, d_flag, num_elems * sizeof(d_t));
        break;
    default:
        std::cerr << "Please specify valid bench type\n";
        exit(1);
        break;
    }

    cuda_err_chk(hipDeviceSynchronize());
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);

    std::cout << "Data: " << (num_elems * sizeof(d_t))/1024/1024/1024 << " gigabytes\n" <<
        "Time: " << ((double)duration.count())/1000000 << " seconds\n" <<
        "Bandwidth: " << ((double)(num_elems * sizeof(d_t)))/1024/1024/1024/((double)duration.count()/1000000) << " GB/s\n";

}
